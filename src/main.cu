#include "hip/hip_runtime.h"
#include <cpptoml.h>
#include <boost/filesystem.hpp>
#include "types.h"
#include "mpm.cuh"
#include "mesh_builder.h"
#include "particle_writer.h"

const unsigned int FrameRate = 240; // for export of data
std::vector<MaterialModel> material_models;
namespace fs = boost::filesystem;

Vec toVector(cpptoml::option<std::vector<double>> &array) {
  Vec out;
  int i = 0;
  for (const double value : *array) {
    out[i] = value;
    i++;
  }
  return out;
}

int main(int argc, char *argv[]) {
  CLIOptions flags(argc, argv);

  Simulation simulation(flags, InterpolationKernel(), material_models);

  fs::path scene_path(flags.scene);
  std::string meshes_dir = fs::system_complete(scene_path.parent_path() / "meshes/").string();

  std::cout << "Loading scene file: " << flags.scene << std::endl;
  auto config = cpptoml::parse_file(flags.scene);
  auto materials = config->get_table_array("material");
  std::map<std::string, u8> material_index;
  int i = 0;
  for (const auto &material : *materials) {
    MaterialModel model(1.0 / flags.particle_count,
                  material->get_as<double>("density").value_or(700.0),
                  material->get_as<double>("E").value_or(1.4e5),
                  material->get_as<double>("Nu").value_or(0.2),
                  material->get_as<double>("hardening").value_or(10.0),
                  material->get_as<double>("plast_clamp_lower").value_or(0.975),
                  material->get_as<double>("plast_clamp_higher").value_or(1.0075));
    std::string name = material->get_as<std::string>("name").value_or("");
    material_index[name] = i;
    material_models.push_back(model);
    i++;
  }

  auto objects = config->get_table_array("object");
  i = 0;
  for (const auto &object : *objects) {
    std::cout << "Adding object " << i << "\r" << std::flush;
    u32 material = material_index[object->get_as<std::string>("material").value_or("")];
    auto position = object->get_array_of<double>("position");
    Vec vec_position = toVector(position);
    auto velocity = object->get_array_of<double>("velocity");
    Vec vec_velocity = toVector(velocity);
    simulation.addObject(meshes_dir + object->get_as<std::string>("mesh").value_or("sphere.obj"),
        material,
        object->get_as<double>("size").value_or(1.0),
        vec_position,
        vec_velocity,
        object->get_as<double>("lifetime_begin").value_or(0.0),
        object->get_as<double>("lifetime_end").value_or(std::numeric_limits<double>::max()));
    i++;
  }

  std::cout << "loaded materials" << std::endl;

  simulation.initCuda();

  ParticleWriter writer;

  Renderer renderer(simulation.getFullParticleCount(), flags.save_dir);

  renderer.render(simulation.getActiveParticleList());

  MeshBuilder mesher(simulation.par, flags, flags.mesh_grid);

  bool save = flags.save_dir != "";
  if (save) {
    fs::create_directory(flags.save_dir);
    std::stringstream ss;
    ss << flags.save_dir << "/meshes";
    fs::create_directory(ss.str());
    ss.str("");
    ss.clear();
    ss << flags.save_dir << "/particles";
    fs::create_directory(ss.str());
  }

  u32 save_every = u32(1. / float(FrameRate) / flags.dt);
  u32 frame_id = 0;
  for (u32 i = 0; i < std::numeric_limits<u32>::max(); i++) {
    if (i % 10 == 0) {
      std::cout << "Step " << i << "\r" << std::flush;
    }
    simulation.advance();
    if (i % 20 == 0) {
      simulation.syncDevice();
      renderer.render(simulation.getActiveParticleList());
    }
    if (save && (i % save_every) == 0) {
      std::stringstream ss;
      ss << flags.save_dir << "/meshes/mesh_" << std::setfill('0') << std::setw(5) << frame_id << ".obj";
      mesher.computeMesh(ss.str(), simulation.getActiveParticleList());
      ss.str("");
      ss.clear();
      ss << flags.save_dir << "/particles/particles_" << frame_id << ".bgeo";
      std::string filepath = ss.str();
      writer.writeParticles(filepath, simulation.getActiveParticleList());
      frame_id++;
    }
  }
}
