#include "hip/hip_runtime.h"
#include "mpm.cuh"
#include "gpu.h"

namespace fs = boost::filesystem;

const real Gravity = -9.81;
const int GridVectorSize = 4;
const int ParticleBlockSize = 64;

float get_random() {
  return float(rand()) / float(RAND_MAX);
}

__global__ void zeroGrid(Vec4* grid) {
  int N = gridDim.x;
  int max_index = N * N * N;
  int base_index = N * N * blockIdx.x + N * threadIdx.x;
  for (int i=0; i < N; i++) {
    int index = base_index +  i;
    if (index >= max_index) return;
    Vec4& cell = grid[index];
    for (int j=0; j < GridVectorSize; j++) {
      cell[j] = 0.0;
    }
  }
}

__global__ void particleToGrid(Particle* particles, Vec4* grid, MaterialModel* material_models,
    int cutoff, SimulationParameters *parameters, InterpolationKernel *interpolation_kernel) {
  __shared__ Particle block_particles[ParticleBlockSize];
  int pi = blockIdx.x * blockDim.x + threadIdx.x;
  float dx = parameters->dx;
  int N = parameters->N;
  if (pi >= cutoff) return;
  block_particles[threadIdx.x] = particles[pi];
  Particle & particle = block_particles[threadIdx.x];
  MaterialModel material_model = material_models[particle.material_type];

  TransferScheme transferScheme;
  transferScheme.p2g_prepare_particle(particle, *parameters, *interpolation_kernel, material_model);

  Veci range_begin = transferScheme.get_range_begin();  // get start of range in grid that is influenced by the particle

  // handle particles that are completely outside of the domain
  for (int i = 0; i < 3; ++i) {
    if (range_begin(i) + int(interpolation_kernel->size()) < 0 || range_begin(i) >= int(N)) {
      return;
    }
  }

  // bounds of particle-influence
  Vec particle_node_distance;
  u32 i_begin = max(0, -range_begin(0));
  u32 j_begin = max(0, -range_begin(1));
  u32 k_begin = max(0, -range_begin(2));
  u32 i_end = min(interpolation_kernel->size(), N + range_begin(0));
  u32 j_end = min(interpolation_kernel->size(), N + range_begin(1));
  u32 k_end = min(interpolation_kernel->size(), N + range_begin(2));

  // loop through relevant grid cells
  for (u32 i = i_begin; i < i_end; ++i) {
    u32 i_glob = range_begin[0] + i;
    u32 index_i = N * N * i_glob;

    particle_node_distance[0] = i_glob * dx - particle.x[0];

    for (u32 j = j_begin; j < j_end; ++j) {
      u32 j_glob = range_begin[1] + j;
      u32 index_j = N * j_glob;
      particle_node_distance[1] = j_glob * dx - particle.x[1];

      for (u32 k = k_begin; k < k_end; ++k) {
        u32 k_glob = range_begin[2] + k;
        particle_node_distance[2] = k_glob * dx - particle.x[2];

        Vec4 node_contribution = transferScheme.p2g_node_contribution(particle, particle_node_distance, material_model.particleMass, i, j, k);

        int index = index_i + index_j + k_glob;
        Vec4& cell = grid[index]; //getCell(i_glob, j_glob, k_glob);
        for (int idx = 0; idx < GridVectorSize; idx++) {
          atomicAdd(&(cell[idx]), node_contribution[idx]);
        }
      }
    }
  }
}

__global__ void gridOpKernel(Vec4* grid, int cutoff, float dt) {
  int N = blockDim.x;
  int x_index = blockIdx.x;
  int y_index = blockIdx.y;
  int z_index = threadIdx.x;
  int index = N * (N * x_index + y_index) + z_index;
  if (index >= N*N*N) return;
  Vec4& cell = grid[index];
  // boundary collisions
  if (cell[3] > 0.0) {
    for (int w=0; w < GridVectorSize; w++) {
      cell[w] /= cell[3];
    }

    cell[1] += dt * Gravity;

    const real boundary = 0.05;

    const real x = real(x_index) / N;
    const real y = real(y_index) / N;
    const real z = real(z_index) / N;
    if (x < boundary || x > 1-boundary || y > 1-boundary || z < boundary || z > 1-boundary) {
      cell[0] = 0.0;
      cell[1] = 0.0;
      cell[2] = 0.0;
      cell[3] = cell[3];
    }
    if (y < boundary) {
      cell[1] = max(real(0.0), cell[1]);
    }
  }
}

__global__ void gridToParticle(Vec4* grid, Particle* particles, MaterialModel* material_models, SimulationParameters &parameters,
    InterpolationKernel &interpolation_kernel) {
  __shared__ Particle block_particles[ParticleBlockSize];
  int pi = blockIdx.x * blockDim.x + threadIdx.x;
  int N = parameters.N;
  if (pi > N * N * N) return;

  block_particles[threadIdx.x] = particles[pi];

  Particle& particle = block_particles[threadIdx.x];
  MaterialModel& material_model = material_models[particle.material_type];

  TransferScheme transferScheme;
  transferScheme.g2p_prepare_particle(particle,
                                      parameters,
                                      interpolation_kernel);

  Veci range_begin = transferScheme.get_range_begin(); // get start of range in grid that is influenced by the particle

  // handle particles that are completely outside of the domain
  for(int i = 0; i < 3; ++i) {
    if(range_begin(i)+int(interpolation_kernel.size()) < 0 || range_begin(i) >= int(N)) {
      return;
    }
  }

  // bounds of particle-influence
  Vec dist_part2node;
  u32 i_begin = max(0, -range_begin(0));
  u32 j_begin = max(0, -range_begin(1));
  u32 k_begin = max(0, -range_begin(2));
  u32 i_end = min(interpolation_kernel.size(), N - range_begin(0));
  u32 j_end = min(interpolation_kernel.size(), N - range_begin(1));
  u32 k_end = min(interpolation_kernel.size(), N - range_begin(2));

  // loop through relevant grid cells
  for(u32 i = i_begin; i < i_end; ++i) {
    u32 i_glob = range_begin(0) + i;
    dist_part2node[0] = i_glob * parameters.dx - particle.x(0);
    u32 index_i = N * N * i_glob;
    for(u32 j = j_begin; j < j_end; ++j) {
      u32 j_glob = range_begin(1) + j;
      dist_part2node[1] = j_glob * parameters.dx - particle.x(1);
      u32 index_j = N * j_glob;

      for(u32 k = k_begin; k < k_end; ++k) {
        u32 k_glob = range_begin(2) + k;
        dist_part2node[2] = k_glob * parameters.dx - particle.x(2);

        // actual transfer
        // velocity
        Vec4& cell = grid[index_i + index_j + k_glob];
        transferScheme.g2p_node_contribution(particle,
                                             dist_part2node,
                                             cell,
                                             i, j, k);
      }
    }
  }

  transferScheme.g2p_finish_particle(particle, parameters);

  // plasticity
  material_model.endOfStepMutation(particle);

  // advection
  particle.x += parameters.dt * particle.v;

  particles[pi] = particle;
}

Simulation::Simulation(const CLIOptions &opts,
           InterpolationKernel const & interpolationKernel, std::vector<MaterialModel> const &material_models) :
    par(opts.dt, opts.N),
    interpolationKernel(interpolationKernel),
    material_models(material_models) {
  N2 = par.N * par.N;
  sizeof_grid = sizeof(Vec4) * N * N * N;
  device_particles = nullptr;
}

Simulation::~Simulation() {
  hipFree(device_grid);
  hipFree(device_particles);
  hipFree(device_material_models);
  delete[] host_particles;
}

void Simulation::initCuda() {
  int bytes = material_models.size() * sizeof(MaterialModel);
  hipMalloc((void **)&device_grid, par.N * par.N * par.N * sizeof(Vec4));
  hipMalloc((void **)&device_material_models, bytes);
  hipMemcpy(device_material_models, material_models.data(), bytes, hipMemcpyHostToDevice);
  hipMalloc((void **)&device_interpolation_kernel, sizeof(InterpolationKernel));
  hipMemcpy(device_interpolation_kernel, &interpolationKernel, sizeof(InterpolationKernel), hipMemcpyHostToDevice);
  hipMalloc((void **)&device_parameters, sizeof(SimulationParameters));
  hipMemcpy(device_parameters, &par, sizeof(SimulationParameters), hipMemcpyHostToDevice);
  particlesToDevice();
}

void Simulation::syncDevice() {
  particlesToHost();
}

void Simulation::resetGrid() {
  zeroGrid<<<int(N), int(N)>>>(device_grid);
}

void Simulation::particleToGridTransfer(std::vector<SimObject> & objects) {
  int n_threads = ParticleBlockSize;
  int grid_width = std::ceil(float(active_particle_count) / float(n_threads));
  particleToGrid<<<grid_width, n_threads>>>(device_particles, device_grid, device_material_models,
      active_particle_count, device_parameters, device_interpolation_kernel);
}

void Simulation::gridOperations() {
  dim3 blocks(N, N);
  int threads = N;
  gridOpKernel<<<blocks, threads>>>(device_grid, N*N*N, par.dt);
}

void Simulation::gridToParticleTransfer(std::vector<SimObject> & objects) {
  int n_threads = ParticleBlockSize;
  int grid_width = std::ceil(float(active_particle_count) / float(n_threads));
  gridToParticle<<<grid_width, n_threads>>>(device_grid, device_particles, device_material_models,
      *device_parameters, *device_interpolation_kernel);
}

void Simulation::addObject(std::string const & filepath,  // obj file defining the shape of the object
               int material_model_index,
               real size,                     // size of the object, measured as the longest edge of the bounding box
               Vec position,                  // position of the object within the scene (lowest corner of object)
               Vec velocity,                  // initial velocity of the object
               real lifetime_begin,
               real lifetime_end) {
  auto mesh = loadMesh(filepath, size, position);
  auto material = material_models[material_model_index];
  auto object = SimObject(material);
  objects.push_back(SimObject(material));
  addParticles(mesh, u32(1.0 / material.particleVolume), velocity, objects.back().particles, material_model_index);
  objects.back().lifetime_begin = lifetime_begin;
  objects.back().lifetime_end = lifetime_end;
}

size_t Simulation::getFullParticleCount() {
  size_t n = 0;
  for(auto & object : objects) {
    n += object.particles.size();
  }
  return n;
}

std::vector<Particle> & Simulation::getFullParticleList() {
  particles_all.resize(0);
  for(auto & object : objects) {
    particles_all.insert(particles_all.end(), object.particles.begin(), object.particles.end());
  }
  return particles_all;
}

std::vector<Particle> & Simulation::getActiveParticleList() {
  particles_all.resize(0);
  for (auto & object : objects) {
    if (!object.isActive(t)) continue;
    particles_all.insert(particles_all.end(), object.particles.begin(), object.particles.end());
  }
  return particles_all;
}

void Simulation::particlesToDevice() {
  std::vector<Particle> &particles = getActiveParticleList();
  if (particles.size() != active_particle_count) {
    active_particle_count = particles.size();
    host_particles = new Particle[active_particle_count];
    hipMalloc((void **)&device_particles, active_particle_count * sizeof(Particle));
  }
  hipMemcpy(device_particles, particles.data(), active_particle_count * sizeof(Particle), hipMemcpyHostToDevice);
}

void Simulation::particlesToHost() {
  hipMemcpy(host_particles, device_particles, active_particle_count * sizeof(Particle), hipMemcpyDeviceToHost);
  int i = 0;
  for (SimObject& object : objects) {
    if (!object.isActive(t)) continue;
    for (int j=0; j < object.particles.size(); j++) {
      object.particles[j] = host_particles[i];
      i++;
      if (i > active_particle_count) {
        active_particle_count = i;
        reallocateParticles();
        return;
      }
    }
  }
  if (i != active_particle_count) {
    std::cout << "Amount of active particles changed" << std::endl;
  }
}

void Simulation::reallocateParticles() {
  hipFree(device_particles);
  delete[] host_particles;
  host_particles = new Particle[active_particle_count];
  hipMalloc((void **)&device_particles, active_particle_count * sizeof(Particle));
  particlesToDevice();
}

void printEnd(const char* name, std::chrono::time_point<std::chrono::high_resolution_clock>& start) {
  auto now = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> diff = now - start;
  std::cout << name << " took " << std::chrono::duration_cast<std::chrono::microseconds>(diff).count() / 10000.0 << "s" << std::endl;
}

#include <chrono>
void Simulation::advance() {
  resetGrid();
  particleToGridTransfer(objects);
  gridOperations();
  gridToParticleTransfer(objects);
  t += par.dt;
}

std::pair<Eigen::MatrixXf, Eigen::MatrixXi> Simulation::loadMesh(const std::string& filepath, double size, Vec position) {
  Eigen::MatrixXf V;
  Eigen::MatrixXi F;
  igl::readOBJ(filepath, V, F);

  Vec min = V.colwise().minCoeff();
  Vec max = V.colwise().maxCoeff();
  real length_max = (max - min).maxCoeff();
  real scale = size / length_max;

  V.array() *= scale;
  V.rowwise() += position.transpose() - scale * min.transpose();

  return std::make_pair(V, F);
}

void Simulation::addParticles(std::pair<Eigen::MatrixXf, Eigen::MatrixXi> const & mesh,
                  u32 particle_density,
                  Vec velocity,
                  std::vector<Particle> & particles,
                  u8 material_index) {
  auto V = std::get<0>(mesh);
  auto F = std::get<1>(mesh);
  Eigen::MatrixXi W;
  const u32 BatchSize = 2048;
  Eigen::Matrix<float, BatchSize, 3> points;
  Vec x;

  int vertices = V.rows();
  double min_x = V.block(0, 0, vertices, 1).minCoeff();
  double max_x = V.block(0, 0, vertices, 1).maxCoeff();
  double min_y = V.block(0, 1, vertices, 1).minCoeff();
  double max_y = V.block(0, 1, vertices, 1).maxCoeff();
  double min_z = V.block(0, 2, vertices, 1).minCoeff();
  double max_z = V.block(0, 2, vertices, 1).maxCoeff();
  double range_x = max_x - min_x;
  double range_y = max_y - min_y;
  double range_z = max_z - min_z;

  double volume_bounding_box = range_x * range_y * range_z;
  u32 particle_count_target = particle_density * volume_bounding_box;

  u32 count_tot = 0;
  while (count_tot < particle_count_target) {
    for (u32 i=0; i < BatchSize; i++) {
      points(i, 0) = min_x + get_random() * range_x;
      points(i, 1) = min_y + get_random() * range_y;
      points(i, 2) = min_z + get_random() * range_z;
    }
    igl::winding_number(V, F, points, W);
    for (u32 i=0; i < BatchSize; i++) {
      count_tot++;
      if (W(i, 0) == 1) {
        x(0) = points(i, 0);
        x(1) = points(i, 1);
        x(2) = points(i, 2);
        particles.push_back(Particle(material_index, x, velocity));
      }
      if (count_tot == particle_count_target) {
        return;
      }
    }
  }
}

