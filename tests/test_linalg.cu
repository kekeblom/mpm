#include "hip/hip_runtime.h"
#include <Eigen/Core>
#include <cmath>
#include "gtest/gtest.h"
#include "types.h"
#include "linalg.h"
#include "gpu.h"

using namespace linalg;

void test_tolerance(const Mat &M, const Mat &R, const Mat &S) {
  Mat M2 = R * S;
  float sum = Eigen::abs((M2 - M).array()).sum();
  ASSERT_LT(sum, 1e-5);
}

void test_tolerance(const Mat &M) {
  Mat R, S;
  polar_decomposition(M, R, S);
  test_tolerance(M, R, S);
  Mat M2 = R * S;
  float sum = Eigen::abs((M2 - M).array()).sum();
  ASSERT_LT(sum, 1e-5);
}

TEST(TestPolar, Basic) {
  Mat M;
  M << 1, 2, 1, 1, 3, 1, 1, 8, 1;
  test_tolerance(M);
}

TEST(TestPolar, Harder) {
  Mat M;
  M << 0, 1, 0, -1, 2, -1, -1, 0.001, -1;
  test_tolerance(M);
}

TEST(TestPolar, UnitaryHermitian) {
  Mat M, R, S;
  M << 1, 2, 1, 1, 3, 1, 1, 8, 1;
  polar_decomposition(M, R, S);
  // Unitary R.
  float diff = Eigen::abs(((R * R.transpose()) - Mat::Identity()).array()).sum();
  ASSERT_LT(diff, 1e-5);
  // Hermitian S.
  diff = Eigen::abs((S - S.transpose()).array()).sum();
  ASSERT_LT(diff, 1e-5);
}

__global__ void polarDecompose(const Mat* A, Mat* R, Mat* S) {
  linalg::polar_decomposition_device(*A, *R, *S);
}

__global__ void computeDeterminant(const Mat& A, real *out) {
  *out = determinant(A);
}

TEST(TestDevicePolar, Basic) {
  Mat M, R, S;
  M << 1, 2, 1, 1, 3, 1, 1, 8, 1;
  Mat * device_M, *device_R, *device_S;
  hipMalloc((void **)&device_M, sizeof(Mat));
  hipMalloc((void **)&device_R, sizeof(Mat));
  hipMalloc((void **)&device_S, sizeof(Mat));
  hipMemcpy(device_M, &M, sizeof(Mat), hipMemcpyHostToDevice);
  polarDecompose<<<1, 1>>>(device_M, device_R, device_S);
  checkGpuError(hipGetLastError());
  checkGpuError(hipDeviceSynchronize());
  hipMemcpy(&R, device_R, sizeof(Mat), hipMemcpyDeviceToHost);
  hipMemcpy(&S, device_S, sizeof(Mat), hipMemcpyDeviceToHost);
  test_tolerance(M, R, S);

  hipFree(device_M);
  hipFree(device_R);
  hipFree(device_S);
}

TEST(TestDeterminant, Identity) {
  Mat M = Mat::Identity();
  Mat * device_M;
  real det;
  real* device_out;
  hipMalloc((void **)&device_M, sizeof(Mat));
  hipMalloc((void **)&device_out, sizeof(real));
  hipMemcpy(device_M, &M, sizeof(Mat), hipMemcpyHostToDevice);
  computeDeterminant<<<1, 1>>>(*device_M, device_out);
  hipMemcpy(&det, device_out, sizeof(real), hipMemcpyDeviceToHost);
  checkGpuError(hipGetLastError());
  checkGpuError(hipDeviceSynchronize());
  ASSERT_EQ(det, 1.0);
  hipFree(device_M);
}

int main(int argc, char **argv) {
  testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}

